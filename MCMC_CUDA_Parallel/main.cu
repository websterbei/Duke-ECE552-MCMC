#include <iostream>
#include <fstream>
#include "metropolis_hastings.h"

using namespace std;

int main(int argc, char* argv[]) {
    const int dimension = 2;
    const int num_samples = argc>1 ? atoi(argv[1]) : 100;
    const int num_blocks = argc>3 ? atoi(argv[2]) : 1;
    const int num_threads = argc>3 ? atoi(argv[3]) : 1;
    // Memory allocation
    float** samples;
    hipMallocManaged(&samples, num_samples*sizeof(float*));
    
    for(int i=0; i<num_samples; i++) {
        hipMallocManaged(&samples[i], dimension*sizeof(float));
    }

    metropolis_hastings<<<num_blocks,num_threads>>>(num_samples, dimension, samples);
    hipDeviceSynchronize();

    ofstream output_file;
    output_file.open("samples.csv");
    for(int i=0; i<num_samples; i++) {
        for(int j=0; j<dimension-1; j++) {
            output_file<<samples[i][j]<<",";
        }
        output_file<<samples[i][dimension-1]<<"\n";
    }
    output_file.close();
    return 0;
}